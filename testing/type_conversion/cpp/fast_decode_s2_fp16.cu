#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 530)
#include <hip/hip_fp16.h>
__device__ half max(half a, half b)
{
  return __hgt(__half(a), __half(b)) ? a : b;
}
__device__ half min(half a, half b)
{
  return __hlt(__half(a), __half(b)) ? a : b;
}
#else

typedef unsigned short uint16_t;
typedef unsigned char uint8_t;
typedef signed char int8_t;
typedef int int32_t;
typedef unsigned long long uint64_t;
typedef unsigned int uint;

#define TVM_FORCE_INLINE inline __attribute__((always_inline))
#define TVM_XINLINE TVM_FORCE_INLINE __device__ __host__
#define TVM_ALIGNED(x) __attribute__((aligned(x)))
#define TVM_HALF_OPERATOR(RTYPE, OP)            \
  TVM_XINLINE RTYPE operator OP(half a, half b) \
  {                                             \
    return RTYPE(float(a) OP float(b));         \
  }                                             \
  template <typename T>                         \
  TVM_XINLINE RTYPE operator OP(half a, T b)    \
  {                                             \
    return RTYPE(float(a) OP float(b));         \
  }                                             \
  template <typename T>                         \
  TVM_XINLINE RTYPE operator OP(T a, half b)    \
  {                                             \
    return RTYPE(float(a) OP float(b));         \
  }

#define TVM_HALF_ASSIGNOP(AOP, OP)                            \
  template <typename T>                                       \
  TVM_XINLINE half operator AOP(const T &a)                   \
  {                                                           \
    return *this = half(float(*this) OP float(a));            \
  }                                                           \
  template <typename T>                                       \
  TVM_XINLINE half operator AOP(const volatile T &a) volatile \
  {                                                           \
    return *this = half(float(*this) OP float(a));            \
  }

class TVM_ALIGNED(2) half
{
public:
  uint16_t half_;

  static TVM_XINLINE half Binary(uint16_t value)
  {
    half res;
    res.half_ = value;
    return res;
  }

  TVM_XINLINE half() {}

  TVM_XINLINE half(const float &value) { constructor(value); }
  TVM_XINLINE explicit half(const double &value) { constructor(value); }
  TVM_XINLINE explicit half(const int8_t &value) { constructor(value); }
  TVM_XINLINE explicit half(const uint8_t &value) { constructor(value); }
  TVM_XINLINE explicit half(const int32_t &value) { constructor(value); }
  TVM_XINLINE explicit half(const uint &value) { constructor(value); }
  TVM_XINLINE explicit half(const long long &value) { constructor(value); }
  TVM_XINLINE explicit half(const uint64_t &value) { constructor(value); }

  TVM_XINLINE operator float() const
  {
    return float(half2float(half_));
  }
  TVM_XINLINE operator float() const volatile
  {
    return float(half2float(half_));
  }

  TVM_HALF_ASSIGNOP(+=, +)
  TVM_HALF_ASSIGNOP(-=, -)
  TVM_HALF_ASSIGNOP(*=, *)
  TVM_HALF_ASSIGNOP(/=, /)

  TVM_XINLINE half operator+()
  {
    return *this;
  }

  TVM_XINLINE half operator-()
  {
    return half(-float(*this));
  }

  TVM_XINLINE half operator=(const half &a)
  {
    half_ = a.half_;
    return a;
  }

  template <typename T>
  TVM_XINLINE half operator=(const T &a)
  {
    return *this = half(a);
  }

  TVM_XINLINE half operator=(const half &a) volatile
  {
    half_ = a.half_;
    return a;
  }

  template <typename T>
  TVM_XINLINE half operator=(const T &a) volatile
  {
    return *this = half(a);
  }

private:
  union Bits
  {
    float f;
    int32_t si;
    uint ui;
  };

  static int const fp16FractionBits = 10;
  static int const fp32FractionBits = 23;
  static int32_t const fp32FractionMask = ~(~0u << fp32FractionBits); // == 0x7fffff
  static int32_t const fp32HiddenBit = 1 << fp32FractionBits;         // == 0x800000
  static int const shift = fp32FractionBits - fp16FractionBits;       // == 13
  static int const shiftSign = 16;
  static int32_t const expAdjust = 127 - 15; // exp32-127 = exp16-15, so exp16 = exp32 - (127-15)

  static int32_t const infN = 0x7F800000;  // flt32 infinity
  static int32_t const maxN = 0x477FFFFF;  // max flt32 that's a flt16 normal after >> by shift
  static int32_t const minN = 0x38800000;  // min flt16 normal as a flt32
  static int32_t const maxZ = 0x33000000;  // max fp32 number that's still rounded to zero in fp16
  static int32_t const signN = 0x80000000; // flt32 sign bit

  static int32_t const infC = infN >> shift;
  static int32_t const nanN = (infC + 1) << shift; // minimum flt16 nan as a flt32
  static int32_t const maxC = maxN >> shift;
  static int32_t const minC = minN >> shift;
  static int32_t const signC = signN >> shiftSign; // flt16 sign bit

  static int32_t const mulN = 0x52000000; // (1 << 23) / minN
  static int32_t const mulC = 0x33800000; // minN / (1 << (23 - shift))

  static int32_t const subC = 0x003FF; // max flt32 subnormal down shifted
  static int32_t const norC = 0x00400; // min flt32 normal down shifted

  static int32_t const maxD = infC - maxC - 1;
  static int32_t const minD = minC - subC - 1;

  TVM_XINLINE uint16_t float2half(const float &value) const
  {
    Bits v;
    v.f = value;
    uint sign = v.si & signN; // grab sign bit
    v.si ^= sign;             // clear sign bit from v
    sign >>= shiftSign;       // logical shift sign to fp16 position

    if (v.si <= maxZ)
    {
      // Handle eventual zeros here to ensure
      // vshift will not exceed 32 below.
      v.ui = 0;
    }
    else if (v.si < minN)
    {
      // Handle denorms
      uint exp32 = v.ui >> fp32FractionBits;
      int32_t exp16 = exp32 - expAdjust;
      // If exp16 == 0 (just into the denorm range), then significant should be shifted right 1.
      // Smaller (so negative) exp16 values should result in greater right shifts.
      uint vshift = 1 - exp16;
      uint significand = fp32HiddenBit | (v.ui & fp32FractionMask);
      v.ui = significand >> vshift;
      v.ui += (v.ui & 0x3fff) != 0x1000 || (significand & 0x7ff) ? 0x1000 : 0;
    }
    else if (v.si <= maxN)
    {
      // Handle norms
      v.ui += (v.ui & 0x3fff) != 0x1000 ? 0x1000 : 0;
      v.ui -= expAdjust << fp32FractionBits;
    }
    else if (v.si <= infN)
    {
      v.si = infN;
    }
    else if (v.si < nanN)
    {
      v.si = nanN;
    }

    v.ui >>= shift;
    return sign | (v.ui & 0x7fff);
  }

  // Same as above routine, except for addition of volatile keyword
  TVM_XINLINE uint16_t float2half(
      const volatile float &value) const volatile
  {
    Bits v;
    v.f = value;
    uint sign = v.si & signN; // grab sign bit
    v.si ^= sign;             // clear sign bit from v
    sign >>= shiftSign;       // logical shift sign to fp16 position

    if (v.si <= maxZ)
    {
      // Handle eventual zeros here to ensure
      // vshift will not exceed 32 below.
      v.ui = 0;
    }
    else if (v.si < minN)
    {
      // Handle denorms
      uint exp32 = v.ui >> fp32FractionBits;
      int32_t exp16 = exp32 - expAdjust;
      // If exp16 == 0 (just into the denorm range), then significant should be shifted right 1.
      // Smaller (so negative) exp16 values should result in greater right shifts.
      uint vshift = 1 - exp16;
      uint significand = fp32HiddenBit | (v.ui & fp32FractionMask);
      v.ui = significand >> vshift;
      v.ui += (v.ui & 0x3fff) != 0x1000 || (significand & 0x7ff) ? 0x1000 : 0;
    }
    else if (v.si <= maxN)
    {
      // Handle norms
      v.ui += (v.ui & 0x3fff) != 0x1000 ? 0x1000 : 0;
      v.ui -= expAdjust << fp32FractionBits;
    }
    else if (v.si <= infN)
    {
      v.si = infN;
    }
    else if (v.si < nanN)
    {
      v.si = nanN;
    }

    v.ui >>= shift;
    return sign | (v.ui & 0x7fff);
  }

  TVM_XINLINE float half2float(const uint16_t &value) const
  {
    Bits v;
    v.ui = value;
    int32_t sign = v.si & signC;
    v.si ^= sign;
    sign <<= shiftSign;
    v.si ^= ((v.si + minD) ^ v.si) & -(v.si > subC);
    v.si ^= ((v.si + maxD) ^ v.si) & -(v.si > maxC);
    Bits s;
    s.si = mulC;
    s.f *= v.si;
    int32_t mask = -(norC > v.si);
    v.si <<= shift;
    v.si ^= (s.si ^ v.si) & mask;
    v.si |= sign;
    return v.f;
  }

  TVM_XINLINE float half2float(
      const volatile uint16_t &value) const volatile
  {
    Bits v;
    v.ui = value;
    int32_t sign = v.si & signC;
    v.si ^= sign;
    sign <<= shiftSign;
    v.si ^= ((v.si + minD) ^ v.si) & -(v.si > subC);
    v.si ^= ((v.si + maxD) ^ v.si) & -(v.si > maxC);
    Bits s;
    s.si = mulC;
    s.f *= v.si;
    int32_t mask = -(norC > v.si);
    v.si <<= shift;
    v.si ^= (s.si ^ v.si) & mask;
    v.si |= sign;
    return v.f;
  }

  template <typename T>
  TVM_XINLINE void constructor(const T &value)
  {
    half_ = float2half(float(value));
  }
};

TVM_HALF_OPERATOR(half, +)
TVM_HALF_OPERATOR(half, -)
TVM_HALF_OPERATOR(half, *)
TVM_HALF_OPERATOR(half, /)
TVM_HALF_OPERATOR(bool, >)
TVM_HALF_OPERATOR(bool, <)
TVM_HALF_OPERATOR(bool, >=)
TVM_HALF_OPERATOR(bool, <=)

TVM_XINLINE half __float2half_rn(const float a)
{
  return half(a);
}
#endif

// Pack two half values.
static inline __device__ __host__ unsigned
__pack_half2(const half x, const half y)
{
  unsigned v0 = *((unsigned short *)&x);
  unsigned v1 = *((unsigned short *)&y);
  return (v1 << 16) | v0;
}

// Some fp16 math functions are not supported in hip/hip_fp16.h,
// so we define them here to make sure the generated CUDA code
// is valid.
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 530)
#define CUDA_UNSUPPORTED_HALF_MATH_BINARY(HALF_MATH_NAME, FP32_MATH_NAME) \
  static inline __device__ __host__ half HALF_MATH_NAME(half x, half y)   \
  {                                                                       \
    float tmp_x = __half2float(x);                                        \
    float tmp_y = __half2float(y);                                        \
    float result = FP32_MATH_NAME(tmp_x, tmp_y);                          \
    return __float2half(result);                                          \
  }

#define CUDA_UNSUPPORTED_HALF_MATH_UNARY(HALF_MATH_NAME, FP32_MATH_NAME) \
  static inline __device__ __host__ half HALF_MATH_NAME(half x)          \
  {                                                                      \
    float tmp_x = __half2float(x);                                       \
    float result = FP32_MATH_NAME(tmp_x);                                \
    return __float2half(result);                                         \
  }

CUDA_UNSUPPORTED_HALF_MATH_BINARY(hpow, powf)
CUDA_UNSUPPORTED_HALF_MATH_UNARY(htanh, tanhf)
CUDA_UNSUPPORTED_HALF_MATH_UNARY(htan, tanf)
CUDA_UNSUPPORTED_HALF_MATH_UNARY(hatan, atanf)
CUDA_UNSUPPORTED_HALF_MATH_UNARY(herf, erf)

#undef CUDA_UNSUPPORTED_HALF_MATH_BINARY
#undef CUDA_UNSUPPORTED_HALF_MATH_UNARY

#endif
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 610)
#include <sm_61_intrinsics.h>
#endif

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ == 800)
#define TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST 1
#else
#define TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST 0
#endif

#ifdef _WIN32
using uint = unsigned int;
using uchar = unsigned char;
using ushort = unsigned short;
using ushort = unsigned short;
using uint64_t = unsigned long long;
#else
#define uint unsigned int
#define uchar unsigned char
#define ushort unsigned short
#define int64_t long long
#define uint64_t unsigned long long
#endif

template <typename T1, typename T2>
__device__ void decode_i2s_to_f16(T1 *_i2s, T2 *B_local_decode, const int N = 16)
{
 uint *h = reinterpret_cast<uint *>(B_local_decode);

  static constexpr uint immLut = (0xf0 & 0xcc) | 0xaa;
  static constexpr uint BOTTOM_MASK = 0x00030003;
  static constexpr uint FP16_TOP_MAGIC_NUM = 0x64006400;
  uint const i2s = *reinterpret_cast<uint *>(_i2s);
  printf("i2s = %x\n", i2s);

#pragma unroll
  // decode 2 elems at one time.
  for (int i = 0; i < (N / 2); i++)
  {

    asm volatile("lop3.b32 %0, %1, %2, %3, %4;\n"
                 : "=r"(h[i])
                 : "r"(i2s >> (2 * i)), "n"(BOTTOM_MASK), "n"(FP16_TOP_MAGIC_NUM), "n"(immLut));
    asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(h[i]) : "r"(h[i]), "r"(FP16_TOP_MAGIC_NUM));
  }
}

extern "C" __global__ void main_kernel0(int8_t *__restrict__ B, half *__restrict__ B_1, const int N = 8)
{
  // print B
  // print B
  for (int i = 0; i < N / 2; i++)
  {
    printf("B[%d] = %d\n", i, (int)B[i]);
  }
  decode_i2s_to_f16(reinterpret_cast<int *>(B), B_1);
  __syncthreads();
  for (int i = 0; i < N; i++)
  {
    printf("B_1[%d] = %f\n", i, float(B_1[i]));
  }
}

void general_compress(const int8_t *lowbit, int8_t *compressed, const int nbit, const int N)
{
  const int nbit_per_byte = 8 / nbit;

  for (int i = 0; i < N / nbit_per_byte; i++)
  {
    for (int j = 0; j < nbit_per_byte; j++)
    {
      compressed[i] |= (lowbit[nbit_per_byte * i + j] << (nbit * j));
    }
  }
}

void general_interleave_fp16_n8(int8_t *origin_arr, int8_t *interleaved, const int nbit, size_t size_in_bytes, bool verbose = false)
{

  // i2s        {e15,e14,e13,e12,e11,e10,e9,e8,e7,e6,e5,e4,e3,e2,e1,e0}
  // interleave {e15,e13,e11,e9,e7,e5,e3,e1,e14,e12,e10,e8,e6,e4,e2,e0}
  // if 16b
  // {e15,e14,e13,e12,e11,e10,e9,e8,e7,e6,e5,e4,e3,e2,e1,e0}
  // Assuming size is the number of int32 elements in origin_arr
  size_t size = size_in_bytes / sizeof(int32_t);
  int32_t *int32_origin = (int32_t *)origin_arr;
  int32_t *int32_interleaved = (int32_t *)interleaved;

  int mask = (1 << nbit) - 1;
  int num_groups = (32 / nbit) / 2;

  for (int idx = 0; idx < size; ++idx)
  {
    int32_t current_value = int32_origin[idx];
    int32_t new_value = 0;

    for (int i = 0; i < num_groups; ++i)
    {
      int left_shift = nbit * i;
      int right_shift = nbit * (num_groups - i - 1);
      new_value |= (current_value & (mask << nbit * (2 * i))) >> left_shift;
      new_value |= (current_value & (mask << nbit * (2 * i + 1))) << right_shift;
      if (verbose)
      {
        printf("put %d to %d\n", (2 * i), (nbit * (2 * i) - left_shift) / nbit);
        printf("put %d to %d\n", (2 * i + 1), (nbit * (2 * i + 1) + right_shift) / nbit);
      }
    }

    int32_interleaved[idx] = new_value;
  }

  // Convert back to int8_t if needed
  memcpy(interleaved, int32_interleaved, size * sizeof(int32_t));
}

int main()
{
  const int nbits = 2;
  // permuate should be done at int32.
  const int N = 32 / nbits;

  // create four int8_t values
  int8_t * lowbit_data = new int8_t[N];
  for (int i = 0; i < N; i++)
  {
    lowbit_data[i] = rand() % 4;
  }
  for (int i = 0; i < N; i++)
  {
    printf("lowbit_data[%d] = %d\n", i, (int)lowbit_data[i]);
  }
  int8_t *is = new int8_t[4];
  general_compress(lowbit_data, is, nbits, N);
  int8_t *interleaved = new int8_t[4];
  general_interleave_fp16_n8(is, interleaved, nbits, 4 * sizeof(int8_t), true);

  half *B_local_decode = new half[N];
  int8_t *is_gpu;
  half *B_local_decode_gpu;

  hipMalloc((void **)&is_gpu, 4 * sizeof(int8_t));
  hipMalloc((void **)&B_local_decode_gpu, N * sizeof(half));
  hipMemcpy(is_gpu, interleaved, 4 * sizeof(int8_t), hipMemcpyHostToDevice);
  hipMemcpy(B_local_decode_gpu, B_local_decode, N * sizeof(half), hipMemcpyHostToDevice);
  // print the last error
  hipError_t cudaerr = hipDeviceSynchronize();
  if (cudaerr != hipSuccess)
    printf("kernel launch failed with error \"%s\".\n",
           hipGetErrorString(cudaerr));
  main_kernel0<<<dim3(1, 1, 1), dim3(1, 1, 1)>>>(is_gpu, B_local_decode_gpu, N);
  // print error
  cudaerr = hipDeviceSynchronize();
  if (cudaerr != hipSuccess)
    printf("kernel launch failed with error \"%s\".\n",
           hipGetErrorString(cudaerr));
  hipMemcpy(B_local_decode, B_local_decode_gpu, N * sizeof(half), hipMemcpyDeviceToHost);
  
  return 0;
}
