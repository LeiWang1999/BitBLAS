#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.
#include <hip/hip_runtime.h>
#include <assert.h>
#include "ladder_kernel.h"
#include "mma.h"
// nvcc ladder_kernel.cu  -gencode arch=compute_80,code=sm_80

__global__ void __launch_bounds__(128) bitblas_kernel_fp16_int2_fp16_m1n15360k5120_nt(half* __restrict__ A, half* __restrict__ QB, half* __restrict__ D) {
            signed char* B = ((int8_t *)QB);
	 half* Scale = (half *)((int8_t *)QB + 19660800); 
	 half* Zeros = (half *)((int8_t *)QB + 20889600);                 
            // const dim3 GridDim(15360, 1, 1);
            // const dim3 BlockDim(128, 1, 1);
            // bitblas_kernel_fp16_int2_fp16_m1n15360k5120_nt<<<GridDim, BlockDim>>>(input_0, input_1, output);
        
  half in_thread_C_local[1];
  signed char B_local[4];
  half B_decode_local[8];
  half A_local[8];
  __shared__ half red_result[1];
  in_thread_C_local[0] = __float2half_rn(0.000000e+00f);
  for (int ax1_0 = 0; ax1_0 < 5; ++ax1_0) {
    *(int*)(B_local + 0) = *(int*)(B + (((((int)blockIdx.x) * 2560) + (ax1_0 * 512)) + (((int)threadIdx.x) * 4)));
    for (int ax1 = 0; ax1 < 8; ++ax1) {
      B_decode_local[ax1] = (((((half)((((uint)B_local[(ax1 >> 1)]) >> (((uint)(ax1 & 1)) * (uint)4)) & (uint)15)) - __float2half_rn(7.000000e+00f)) * Scale[(((((int)blockIdx.x) * 40) + (ax1_0 * 8)) + (((int)threadIdx.x) >> 4))]) - Zeros[(((((int)blockIdx.x) * 40) + (ax1_0 * 8)) + (((int)threadIdx.x) >> 4))]);
    }
    *(uint4*)(A_local + 0) = *(uint4*)(A + ((ax1_0 * 1024) + (((int)threadIdx.x) * 8)));
    for (int ax1_2_0 = 0; ax1_2_0 < 4; ++ax1_2_0) {
      for (int ax1_2_1 = 0; ax1_2_1 < 2; ++ax1_2_1) {
        in_thread_C_local[0] = (in_thread_C_local[0] + (A_local[((ax1_2_0 * 2) + ax1_2_1)] * B_decode_local[((ax1_2_0 * 2) + ax1_2_1)]));
      }
    }
  }
  half red_buf0[1];
  uint mask[1];
  half t0[1];
  half red_buf0_1[1];
  uint mask_1[1];
  half t0_1[1];
  __shared__ half red_buf_staging[4];
  red_buf0_1[0] = in_thread_C_local[0];
  mask_1[0] = __activemask();
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 16, 32);
  red_buf0_1[0] = (red_buf0_1[0] + t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 8, 32);
  red_buf0_1[0] = (red_buf0_1[0] + t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 4, 32);
  red_buf0_1[0] = (red_buf0_1[0] + t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 2, 32);
  red_buf0_1[0] = (red_buf0_1[0] + t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 1, 32);
  red_buf0_1[0] = (red_buf0_1[0] + t0_1[0]);
  if ((((int)threadIdx.x) % 32) == 0) {
    red_buf_staging[(((int)threadIdx.x) >> 5)] = red_buf0_1[0];
  }
  __syncthreads();
  if (((int)threadIdx.x) < 4) {
    red_buf0[0] = red_buf_staging[((int)threadIdx.x)];
  }
  mask[0] = (__activemask() & (uint)15);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 2, 32);
  red_buf0[0] = (red_buf0[0] + t0[0]);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 1, 32);
  red_buf0[0] = (red_buf0[0] + t0[0]);
  if (((int)threadIdx.x) == 0) {
    ((volatile half*)red_result)[0] = red_buf0[0];
  }
  __syncthreads();
  if (((int)threadIdx.x) == 0) {
    D[((int)blockIdx.x)] = (half)(((volatile half*)red_result)[0]);
  }
}



__global__ void __launch_bounds__(128) bitblas_kernel_fp16_int2_fp16_m128n15360k5120_nt(half* __restrict__ A, half* __restrict__ QB, half* __restrict__ D) {
            signed char* B = ((int8_t *)QB);
	 half* Scale = (half *)((int8_t *)QB + 19660800); 
	 half* Zeros = (half *)((int8_t *)QB + 20889600);                 
            // const dim3 GridDim(160, 2, 1);
            // const dim3 BlockDim(32, 4, 1);
            // bitblas_kernel_fp16_int2_fp16_m128n15360k5120_nt<<<GridDim, BlockDim>>>(input_0, input_1, output);
        

        const int MAX_BLOCK_N = 10;
        const auto baseBlockIdx = blockIdx.x + gridDim.x *blockIdx.y;
        const auto totalPanel = (gridDim.x * gridDim.y +MAX_BLOCK_N * gridDim.x - 1) / (MAX_BLOCK_N * gridDim.x);
        const auto totalBlock = gridDim.x * gridDim.y;
        const auto panelIdx = baseBlockIdx / (MAX_BLOCK_N *gridDim.x);
        const auto strideLd = panelIdx + 1 < totalPanel ?MAX_BLOCK_N : (totalBlock - panelIdx * (MAX_BLOCK_N *gridDim.x)) / gridDim.x;
        const auto bx = (panelIdx & 1) ? gridDim.x -(baseBlockIdx - panelIdx * MAX_BLOCK_N * gridDim.x) /strideLd - 1 : (baseBlockIdx - panelIdx * MAX_BLOCK_N *gridDim.x) / strideLd;
        const auto by = (baseBlockIdx - panelIdx * MAX_BLOCK_N *gridDim.x) % strideLd + panelIdx * MAX_BLOCK_N;
        const auto bz = blockIdx.z;
        const dim3 blockIdx(bx, by, bz);
      half C_reindex_shared_warp[48];
  __shared__ half A_reindex_shared[4096];
  __shared__ signed char B_shared[3072];
  __shared__ half B_decode_reindex_shared[3072];
  signed char B_local[4];
  half B_decode_reindex_local[8];
  half A_reindex_shared_warp[8];
  half B_decode_reindex_shared_warp[48];
  signed char B_local_1[4];
  half B_decode_reindex_local_1[8];
  half A_reindex_shared_warp_1[8];
  half B_decode_reindex_shared_warp_1[48];
  for (int var = 0; var < 1; ++var) {
    for (int ax2_0_3_init = 0; ax2_0_3_init < 6; ++ax2_0_3_init) {
      for (int i = 0; i < 8; ++i) {
C_reindex_shared_warp[(ax2_0_3_init * 8) + i] = 0.0;}
;
    }
    #pragma unroll
    for (int ax0_ax1_ax2_fused_0 = 0; ax0_ax1_ax2_fused_0 < 2; ++ax0_ax1_ax2_fused_0) {

  {
        unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
    addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)(A_reindex_shared + ((((ax0_ax1_ax2_fused_0 * 1024) + (((int)threadIdx.y) * 256)) + ((((int)threadIdx.x) >> 2) * 32)) + (((((int)threadIdx.x) & 3) ^ (((int)threadIdx.x) >> 3)) * 8)))));
#else
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_reindex_shared + ((((ax0_ax1_ax2_fused_0 * 1024) + (((int)threadIdx.y) * 256)) + ((((int)threadIdx.x) >> 2) * 32)) + (((((int)threadIdx.x) & 3) ^ (((int)threadIdx.x) >> 3)) * 8))))
    );
#endif
    __asm__ __volatile__(
      #if TVM_ENABLE_L2_PREFETCH
        "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;"
      #else
        "cp.async.cg.shared.global [%0], [%1], %2;"
      #endif
        :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.y) * 327680) + (ax0_ax1_ax2_fused_0 * 163840)) + (((int)threadIdx.y) * 40960)) + ((((int)threadIdx.x) >> 2) * 5120)) + ((((int)threadIdx.x) & 3) * 8)))), "n"(16)
    );
  }
    }
    __syncthreads();
    #pragma unroll
    for (int ax0_ax1_fused_0 = 0; ax0_ax1_fused_0 < 1; ++ax0_ax1_fused_0) {
      if (((int)threadIdx.y) < 3) {

  {
        unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
    addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)(B_shared + ((((int)threadIdx.y) * 512) + (((int)threadIdx.x) * 16)))));
#else
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.y) * 512) + (((int)threadIdx.x) * 16))))
    );
#endif
    __asm__ __volatile__(
      #if TVM_ENABLE_L2_PREFETCH
        "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;"
      #else
        "cp.async.cg.shared.global [%0], [%1], %2;"
      #endif
        :: "r"(addr), "l"((void*)(B + (((((int)blockIdx.x) * 245760) + (((int)threadIdx.y) * 81920)) + (((int)threadIdx.x) * 2560)))), "n"(16)
    );
  }
      }
    }
__asm__ __volatile__("cp.async.commit_group;");

    for (int ax3_0_0 = 0; ax3_0_0 < 159; ++ax3_0_0) {
      __syncthreads();
      #pragma unroll
      for (int ax0_ax1_ax2_fused_0_1 = 0; ax0_ax1_ax2_fused_0_1 < 2; ++ax0_ax1_ax2_fused_0_1) {

  {
        unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
    addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)(A_reindex_shared + (((((((ax3_0_0 + 1) & 1) * 2048) + (ax0_ax1_ax2_fused_0_1 * 1024)) + (((int)threadIdx.y) * 256)) + ((((int)threadIdx.x) >> 2) * 32)) + (((((int)threadIdx.x) & 3) ^ (((int)threadIdx.x) >> 3)) * 8)))));
#else
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_reindex_shared + (((((((ax3_0_0 + 1) & 1) * 2048) + (ax0_ax1_ax2_fused_0_1 * 1024)) + (((int)threadIdx.y) * 256)) + ((((int)threadIdx.x) >> 2) * 32)) + (((((int)threadIdx.x) & 3) ^ (((int)threadIdx.x) >> 3)) * 8))))
    );
#endif
    __asm__ __volatile__(
      #if TVM_ENABLE_L2_PREFETCH
        "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;"
      #else
        "cp.async.cg.shared.global [%0], [%1], %2;"
      #endif
        :: "r"(addr), "l"((void*)(A + (((((((((int)blockIdx.y) * 327680) + (ax0_ax1_ax2_fused_0_1 * 163840)) + (((int)threadIdx.y) * 40960)) + ((((int)threadIdx.x) >> 2) * 5120)) + (ax3_0_0 * 32)) + ((((int)threadIdx.x) & 3) * 8)) + 32))), "n"(16)
    );
  }
      }
      #pragma unroll
      for (int ax0_ax1_fused_0_1 = 0; ax0_ax1_fused_0_1 < 1; ++ax0_ax1_fused_0_1) {
        if (((int)threadIdx.y) < 3) {

  {
        unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
    addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)(B_shared + (((((ax3_0_0 + 1) & 1) * 1536) + (((int)threadIdx.y) * 512)) + (((int)threadIdx.x) * 16)))));
#else
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + (((((ax3_0_0 + 1) & 1) * 1536) + (((int)threadIdx.y) * 512)) + (((int)threadIdx.x) * 16))))
    );
#endif
    __asm__ __volatile__(
      #if TVM_ENABLE_L2_PREFETCH
        "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;"
      #else
        "cp.async.cg.shared.global [%0], [%1], %2;"
      #endif
        :: "r"(addr), "l"((void*)(B + (((((((int)blockIdx.x) * 245760) + (((int)threadIdx.y) * 81920)) + (((int)threadIdx.x) * 2560)) + (ax3_0_0 * 16)) + 16))), "n"(16)
    );
  }
        }
      }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 1;");

      __syncthreads();
      for (int ax1_ax2_0_fused_0 = 0; ax1_ax2_0_fused_0 < 3; ++ax1_ax2_0_fused_0) {
        *(int*)(B_local + 0) = *(int*)(B_shared + (((((ax3_0_0 & 1) * 1536) + (ax1_ax2_0_fused_0 * 512)) + (((int)threadIdx.y) * 128)) + (((int)threadIdx.x) * 4)));
        for (int ax2 = 0; ax2 < 8; ++ax2) {
          B_decode_reindex_local[ax2] = (((((half)((((uint)B_local[(ax2 >> 1)]) >> (((uint)(ax2 & 1)) * (uint)4)) & (uint)15)) - __float2half_rn(7.000000e+00f)) * Scale[(((((((int)blockIdx.x) * 3840) + (ax1_ax2_0_fused_0 * 1280)) + (((int)threadIdx.y) * 320)) + ((((int)threadIdx.x) >> 2) * 40)) + (ax3_0_0 >> 2))]) - Zeros[(((((((int)blockIdx.x) * 3840) + (ax1_ax2_0_fused_0 * 1280)) + (((int)threadIdx.y) * 320)) + ((((int)threadIdx.x) >> 2) * 40)) + (ax3_0_0 >> 2))]);
        }
        *(uint4*)(B_decode_reindex_shared + ((((ax1_ax2_0_fused_0 * 1024) + (((int)threadIdx.y) * 256)) + ((((int)threadIdx.x) >> 2) * 32)) + (((((int)threadIdx.x) & 3) ^ (((int)threadIdx.x) >> 3)) * 8))) = *(uint4*)(B_decode_reindex_local + 0);
      }
      __syncthreads();
      for (int ax3_0_1 = 0; ax3_0_1 < 2; ++ax3_0_1) {

  {
    unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
    addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)((&(A_reindex_shared[(((((ax3_0_0 & 1) * 2048) + (((int)threadIdx.y) * 512)) + ((((int)threadIdx.x) & 15) * 32)) + ((((ax3_0_1 * 2) + (((int)threadIdx.x) >> 4)) ^ ((((int)threadIdx.x) & 7) >> 1)) * 8))])) + 0)));
#else
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)((&(A_reindex_shared[(((((ax3_0_0 & 1) * 2048) + (((int)threadIdx.y) * 512)) + ((((int)threadIdx.x) & 15) * 32)) + ((((ax3_0_1 * 2) + (((int)threadIdx.x) >> 4)) ^ ((((int)threadIdx.x) & 7) >> 1)) * 8))])) + 0))
    );
#endif
    __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
      "{%0, %1, %2, %3}, [%4];\n"
      : "=r"(((unsigned *)(A_reindex_shared_warp + 0))[0]), "=r"(((unsigned *)(A_reindex_shared_warp + 0))[1]), "=r"(((unsigned *)(A_reindex_shared_warp + 0))[2]), "=r"(((unsigned *)(A_reindex_shared_warp + 0))[3])
      : "r"(addr)
    );
  }
        for (int ax1_0 = 0; ax1_0 < 6; ++ax1_0) {

  {
    unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
    addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)((&(B_decode_reindex_shared[((((ax1_0 * 512) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 7) * 32)) + ((((ax3_0_1 * 2) + ((((int)threadIdx.x) & 15) >> 3)) ^ ((((int)threadIdx.x) & 7) >> 1)) * 8))])) + 0)));
#else
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)((&(B_decode_reindex_shared[((((ax1_0 * 512) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 7) * 32)) + ((((ax3_0_1 * 2) + ((((int)threadIdx.x) & 15) >> 3)) ^ ((((int)threadIdx.x) & 7) >> 1)) * 8))])) + 0))
    );
#endif
    __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
      "{%0, %1, %2, %3}, [%4];\n"
      : "=r"(((unsigned *)(B_decode_reindex_shared_warp + (ax1_0 * 8)))[0]), "=r"(((unsigned *)(B_decode_reindex_shared_warp + (ax1_0 * 8)))[1]), "=r"(((unsigned *)(B_decode_reindex_shared_warp + (ax1_0 * 8)))[2]), "=r"(((unsigned *)(B_decode_reindex_shared_warp + (ax1_0 * 8)))[3])
      : "r"(addr)
    );
  }
        }
        for (int ax2_0_3 = 0; ax2_0_3 < 6; ++ax2_0_3) {

  {
    __asm__ __volatile__(
      "mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16"
      "{%0, %1}, {%2, %3, %4, %5}, {%6, %7}, {%8, %9};\n"
      :  "=r"(((unsigned *)(C_reindex_shared_warp + (ax2_0_3 * 8)))[0]), "=r"(((unsigned *)(C_reindex_shared_warp + (ax2_0_3 * 8)))[1])
      : "r"(((unsigned *)(A_reindex_shared_warp + 0))[0]), "r"(((unsigned *)(A_reindex_shared_warp + 0))[1]), "r"(((unsigned *)(A_reindex_shared_warp + 0))[2]), "r"(((unsigned *)(A_reindex_shared_warp + 0))[3]), "r"(((unsigned *)(B_decode_reindex_shared_warp + (ax2_0_3 * 8)))[0]), "r"(((unsigned *)(B_decode_reindex_shared_warp + (ax2_0_3 * 8)))[1]), "r"(((unsigned *)(C_reindex_shared_warp + (ax2_0_3 * 8)))[0]), "r"(((unsigned *)(C_reindex_shared_warp + (ax2_0_3 * 8)))[1]));
  }

  {
    __asm__ __volatile__(
      "mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16"
      "{%0, %1}, {%2, %3, %4, %5}, {%6, %7}, {%8, %9};\n"
      :  "=r"(((unsigned *)(C_reindex_shared_warp + ((ax2_0_3 * 8) + 4)))[0]), "=r"(((unsigned *)(C_reindex_shared_warp + ((ax2_0_3 * 8) + 4)))[1])
      : "r"(((unsigned *)(A_reindex_shared_warp + 0))[0]), "r"(((unsigned *)(A_reindex_shared_warp + 0))[1]), "r"(((unsigned *)(A_reindex_shared_warp + 0))[2]), "r"(((unsigned *)(A_reindex_shared_warp + 0))[3]), "r"(((unsigned *)(B_decode_reindex_shared_warp + ((ax2_0_3 * 8) + 4)))[0]), "r"(((unsigned *)(B_decode_reindex_shared_warp + ((ax2_0_3 * 8) + 4)))[1]), "r"(((unsigned *)(C_reindex_shared_warp + ((ax2_0_3 * 8) + 4)))[0]), "r"(((unsigned *)(C_reindex_shared_warp + ((ax2_0_3 * 8) + 4)))[1]));
  }
        }
      }
    }
__asm__ __volatile__("cp.async.wait_group 0;");

    __syncthreads();
    for (int ax1_ax2_0_fused_0_1 = 0; ax1_ax2_0_fused_0_1 < 3; ++ax1_ax2_0_fused_0_1) {
      *(int*)(B_local_1 + 0) = *(int*)(B_shared + ((((ax1_ax2_0_fused_0_1 * 512) + (((int)threadIdx.y) * 128)) + (((int)threadIdx.x) * 4)) + 1536));
      for (int ax2_1 = 0; ax2_1 < 8; ++ax2_1) {
        B_decode_reindex_local_1[ax2_1] = (((((half)((((uint)B_local_1[(ax2_1 >> 1)]) >> (((uint)(ax2_1 & 1)) * (uint)4)) & (uint)15)) - __float2half_rn(7.000000e+00f)) * Scale[(((((((int)blockIdx.x) * 3840) + (ax1_ax2_0_fused_0_1 * 1280)) + (((int)threadIdx.y) * 320)) + ((((int)threadIdx.x) >> 2) * 40)) + 39)]) - Zeros[(((((((int)blockIdx.x) * 3840) + (ax1_ax2_0_fused_0_1 * 1280)) + (((int)threadIdx.y) * 320)) + ((((int)threadIdx.x) >> 2) * 40)) + 39)]);
      }
      *(uint4*)(B_decode_reindex_shared + ((((ax1_ax2_0_fused_0_1 * 1024) + (((int)threadIdx.y) * 256)) + ((((int)threadIdx.x) >> 2) * 32)) + (((((int)threadIdx.x) & 3) ^ (((int)threadIdx.x) >> 3)) * 8))) = *(uint4*)(B_decode_reindex_local_1 + 0);
    }
    __syncthreads();
    for (int ax3_0_1_1 = 0; ax3_0_1_1 < 2; ++ax3_0_1_1) {

  {
    unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
    addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)((&(A_reindex_shared[((((((int)threadIdx.y) * 512) + ((((int)threadIdx.x) & 15) * 32)) + ((((ax3_0_1_1 * 2) + (((int)threadIdx.x) >> 4)) ^ ((((int)threadIdx.x) & 7) >> 1)) * 8)) + 2048)])) + 0)));
#else
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)((&(A_reindex_shared[((((((int)threadIdx.y) * 512) + ((((int)threadIdx.x) & 15) * 32)) + ((((ax3_0_1_1 * 2) + (((int)threadIdx.x) >> 4)) ^ ((((int)threadIdx.x) & 7) >> 1)) * 8)) + 2048)])) + 0))
    );
#endif
    __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
      "{%0, %1, %2, %3}, [%4];\n"
      : "=r"(((unsigned *)(A_reindex_shared_warp_1 + 0))[0]), "=r"(((unsigned *)(A_reindex_shared_warp_1 + 0))[1]), "=r"(((unsigned *)(A_reindex_shared_warp_1 + 0))[2]), "=r"(((unsigned *)(A_reindex_shared_warp_1 + 0))[3])
      : "r"(addr)
    );
  }
      for (int ax1_0_1 = 0; ax1_0_1 < 6; ++ax1_0_1) {

  {
    unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
    addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)((&(B_decode_reindex_shared[((((ax1_0_1 * 512) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 7) * 32)) + ((((ax3_0_1_1 * 2) + ((((int)threadIdx.x) & 15) >> 3)) ^ ((((int)threadIdx.x) & 7) >> 1)) * 8))])) + 0)));
#else
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)((&(B_decode_reindex_shared[((((ax1_0_1 * 512) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 7) * 32)) + ((((ax3_0_1_1 * 2) + ((((int)threadIdx.x) & 15) >> 3)) ^ ((((int)threadIdx.x) & 7) >> 1)) * 8))])) + 0))
    );
#endif
    __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
      "{%0, %1, %2, %3}, [%4];\n"
      : "=r"(((unsigned *)(B_decode_reindex_shared_warp_1 + (ax1_0_1 * 8)))[0]), "=r"(((unsigned *)(B_decode_reindex_shared_warp_1 + (ax1_0_1 * 8)))[1]), "=r"(((unsigned *)(B_decode_reindex_shared_warp_1 + (ax1_0_1 * 8)))[2]), "=r"(((unsigned *)(B_decode_reindex_shared_warp_1 + (ax1_0_1 * 8)))[3])
      : "r"(addr)
    );
  }
      }
      for (int ax2_0_3_1 = 0; ax2_0_3_1 < 6; ++ax2_0_3_1) {

  {
    __asm__ __volatile__(
      "mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16"
      "{%0, %1}, {%2, %3, %4, %5}, {%6, %7}, {%8, %9};\n"
      :  "=r"(((unsigned *)(C_reindex_shared_warp + (ax2_0_3_1 * 8)))[0]), "=r"(((unsigned *)(C_reindex_shared_warp + (ax2_0_3_1 * 8)))[1])
      : "r"(((unsigned *)(A_reindex_shared_warp_1 + 0))[0]), "r"(((unsigned *)(A_reindex_shared_warp_1 + 0))[1]), "r"(((unsigned *)(A_reindex_shared_warp_1 + 0))[2]), "r"(((unsigned *)(A_reindex_shared_warp_1 + 0))[3]), "r"(((unsigned *)(B_decode_reindex_shared_warp_1 + (ax2_0_3_1 * 8)))[0]), "r"(((unsigned *)(B_decode_reindex_shared_warp_1 + (ax2_0_3_1 * 8)))[1]), "r"(((unsigned *)(C_reindex_shared_warp + (ax2_0_3_1 * 8)))[0]), "r"(((unsigned *)(C_reindex_shared_warp + (ax2_0_3_1 * 8)))[1]));
  }

  {
    __asm__ __volatile__(
      "mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16"
      "{%0, %1}, {%2, %3, %4, %5}, {%6, %7}, {%8, %9};\n"
      :  "=r"(((unsigned *)(C_reindex_shared_warp + ((ax2_0_3_1 * 8) + 4)))[0]), "=r"(((unsigned *)(C_reindex_shared_warp + ((ax2_0_3_1 * 8) + 4)))[1])
      : "r"(((unsigned *)(A_reindex_shared_warp_1 + 0))[0]), "r"(((unsigned *)(A_reindex_shared_warp_1 + 0))[1]), "r"(((unsigned *)(A_reindex_shared_warp_1 + 0))[2]), "r"(((unsigned *)(A_reindex_shared_warp_1 + 0))[3]), "r"(((unsigned *)(B_decode_reindex_shared_warp_1 + ((ax2_0_3_1 * 8) + 4)))[0]), "r"(((unsigned *)(B_decode_reindex_shared_warp_1 + ((ax2_0_3_1 * 8) + 4)))[1]), "r"(((unsigned *)(C_reindex_shared_warp + ((ax2_0_3_1 * 8) + 4)))[0]), "r"(((unsigned *)(C_reindex_shared_warp + ((ax2_0_3_1 * 8) + 4)))[1]));
  }
      }
    }
    for (int ax0 = 0; ax0 < 6; ++ax0) {
      __syncthreads();
      for (int local_id = 0; local_id < 8; local_id+=2) {
*((uint *)&(&(((half*)B_shared)[(((int)threadIdx.y) * 256)]))[((((((local_id % 4) / 2) * 8) + (threadIdx.x / 4)) * 16) + ((((local_id / 4) * 8) + ((threadIdx.x % 4) * 2)) + (local_id % 2)))]) = *((uint *)&C_reindex_shared_warp[(ax0 * 8) + local_id]);
}
;
      __syncthreads();
      #pragma unroll
      for (int ax0_ax1_ax2_ax3_ax4_fused_0 = 0; ax0_ax1_ax2_ax3_ax4_fused_0 < 1; ++ax0_ax1_ax2_ax3_ax4_fused_0) {
        *(uint4*)(D + ((((((((int)blockIdx.y) * 983040) + (((int)threadIdx.y) * 245760)) + ((((int)threadIdx.x) >> 1) * 15360)) + (((int)blockIdx.x) * 96)) + (ax0 * 16)) + ((((int)threadIdx.x) & 1) * 8))) = *(uint4*)(((half*)B_shared) + ((((int)threadIdx.y) * 256) + (((int)threadIdx.x) * 8)));
      }
    }
  }
}





int ladder_gemm_fp16xint2_fp16(half *input_0, half *input_1, half *output, const int M, const int N, const int K, const int trans_a, const int trans_b, half *workspace_ptr)
{
    assert(trans_a == 0 && trans_b == 1);
    
    if (M == 1 && N == 15360 && K == 5120){
        
             const dim3 GridDim(15360, 1, 1);
             const dim3 BlockDim(128, 1, 1);
             bitblas_kernel_fp16_int2_fp16_m1n15360k5120_nt<<<GridDim, BlockDim>>>(input_0, input_1, output);
        
        return 0;
    }

    
    if (M == 128 && N == 15360 && K == 5120){
        
             const dim3 GridDim(160, 2, 1);
             const dim3 BlockDim(32, 4, 1);
             bitblas_kernel_fp16_int2_fp16_m128n15360k5120_nt<<<GridDim, BlockDim>>>(input_0, input_1, output);
        
        return 0;
    }

    
    return -1;
}